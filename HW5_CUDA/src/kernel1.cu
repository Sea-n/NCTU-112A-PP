#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>

__device__ int mandel(float c_re, float c_im, int count) {
  float z_re = c_re, z_im = c_im;
  int i;
  for (i = 0; i < count; ++i) {
    if (z_re * z_re + z_im * z_im > 4.f)
      break;

    float new_re = z_re * z_re - z_im * z_im;
    float new_im = 2.f * z_re * z_im;
    z_re = c_re + new_re;
    z_im = c_im + new_im;
  }

  return i;
}

__global__ void mandelKernel(int* img_dev, float x0, float dx, int resX,
		float y0, float dy, int resY, int maxIterations) {
    int thisX = blockIdx.x * blockDim.x + threadIdx.x;
    int thisY = blockIdx.y * blockDim.y + threadIdx.y;

    float x = x0 + thisX * dx;
    float y = y0 + thisY * dy;

    int idx = resX * thisY + thisX;
    img_dev[idx] = mandel(x, y, maxIterations);
}

void hostFE(float upperX, float upperY, float lowerX, float lowerY, int* img, int resX, int resY, int maxIterations) {
    float dx = (upperX - lowerX) / resX;
    float dy = (upperY - lowerY) / resY;

    int size = resX * resY * sizeof(int);
    int *img_host = (int *) malloc(size * sizeof(int));
    if (!img_host) {
        free(img_host);
        return;
    }

    int *img_dev;
    hipMalloc(&img_dev, size);

    int block_x = 32;
    int block_y = 32;
    int grid_x = (int) ceil(resX / (float) block_x);
    int grid_y = (int) ceil(resY / (float) block_y);

    dim3 dimBlock(block_x, block_y);
    dim3 dimGrid(grid_x, grid_y);
    mandelKernel<<<dimGrid, dimBlock>>>(img_dev, lowerX, dx, resX, lowerY, dy, resY, maxIterations);

    hipMemcpy(img_host, img_dev, size, hipMemcpyDeviceToHost);
    memcpy(img, img_host, size);
    free(img_host);
    hipFree(img_dev);
}
