#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>

__device__ int mandel(float c_re, float c_im, int count) {
  float z_re = c_re, z_im = c_im;
  int i;
  for (i = 0; i < count; ++i) {
    if (z_re * z_re + z_im * z_im > 4.f)
      break;

    float new_re = z_re * z_re - z_im * z_im;
    float new_im = 2.f * z_re * z_im;
    z_re = c_re + new_re;
    z_im = c_im + new_im;
  }

  return i;
}

__global__ void mandelKernel(int* img_dev, float x0, float dx, int resX,
		float y0, float dy, int resY, int maxIterations) {
    for (int i=0; i<4; i++) {
	    int thisX = blockIdx.x * blockDim.x + threadIdx.x;
	    int thisY = (blockIdx.y * blockDim.y + threadIdx.y) * 4 + i;

	    float x = x0 + thisX * dx;
	    float y = y0 + thisY * dy;

	    int idx = resX * thisY + thisX;
	    img_dev[idx] = mandel(x, y, maxIterations);
    }
}

void hostFE(float upperX, float upperY, float lowerX, float lowerY, int* img, int resX, int resY, int maxIterations) {
    float dx = (upperX - lowerX) / resX;
    float dy = (upperY - lowerY) / resY;

    int size = resX * resY * sizeof(int);
    int *img_host = (int *) malloc(size * sizeof(int));
    if (!img_host) {
        free(img_host);
        return;
    }
    hipHostAlloc(&img_host, size, hipHostMallocMapped);

    int *img_dev;
    size_t pitch;
    hipMallocPitch(&img_dev, &pitch, resX * sizeof(int), resY);

    int block_x = 32;
    int block_y = 32;
    int grid_x = (int) ceil(resX / (float) block_x);
    int grid_y = (int) ceil(resY / (float) block_y / 4);

    dim3 dimBlock(block_x, block_y);
    dim3 dimGrid(grid_x, grid_y);
    mandelKernel<<<dimGrid, dimBlock>>>(img_dev, lowerX, dx, resX, lowerY, dy, resY, maxIterations);

    hipDeviceSynchronize();

    hipMemcpy(img_host, img_dev, size, hipMemcpyDeviceToHost);
    memcpy(img, img_host, size);
    hipHostFree(img_host);
    hipFree(img_dev);
}
